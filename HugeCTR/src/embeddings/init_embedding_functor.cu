#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "HugeCTR/include/data_simulator.hpp"
#include "HugeCTR/include/embeddings/sparse_embedding_functors.hpp"
#include "HugeCTR/include/general_buffer2.hpp"
#include "HugeCTR/include/utils.hpp"

namespace HugeCTR {
/*
我们来分析 init_embedding_per_gpu，其实就是简单的用 % 运算来进行分配。
 举出一个例子来看看：假如10个slot，3个GPU，则slot ID是 0～9，
 GPU id是0～2。0~10 % 3 = 0,1,2,0,1,2,0,1,2,0，
 所以10个slot 被分配到3个GPU，分别是：
    GPU 0 ：0，3，6，9
    GPU 1 : 1，4，7，
    GPU 2 ：2，5，8，
所以，slot per gpu 是不相等的。
 */
void SparseEmbeddingFunctors::init_embedding_per_gpu(size_t gid, size_t total_gpu_count,
                                                     const std::vector<size_t> &slot_sizes,
                                                     size_t embedding_vec_size,
                                                     Tensors2<float> &embedding_tables,
                                                     Tensor2<size_t> &slot_ids,
                                                     const GPUResource &gpu_resource) {
  CudaDeviceContext context(gpu_resource.get_device_id());

  size_t *slot_ids_ptr = slot_ids.get_ptr();

  size_t key_offset = 0;
  size_t value_index_offset = 0;
  for (size_t i = 0, j = 0; i < slot_sizes.size(); i++) {  // 遍历slot
    size_t slot_size = slot_sizes[i];
    if ((i % total_gpu_count) == gid) {  // 本GPU id
      MESSAGE_("gpu" + std::to_string(gid) + " start to init embedding of slot" +
               std::to_string(i) + " , slot_size=" + std::to_string(slot_size) +
               ", key_offset=" + std::to_string(key_offset) +
               ", value_index_offset=" + std::to_string(value_index_offset));

      // 只有i等于gid时候，才会继续操作
      float up_bound = sqrt(1.f / slot_size);
      HugeCTR::UniformGenerator::fill(
          embedding_tables[j++], -up_bound, up_bound, gpu_resource.get_sm_count(),
          gpu_resource.get_replica_variant_curand_generator(), gpu_resource.get_stream());

      // 配置slot id
      memset_const(slot_ids_ptr, i, slot_size, gpu_resource.get_stream());

      value_index_offset += slot_size;
      slot_ids_ptr += slot_size;
    }
    key_offset += slot_size;
  }
}

}  // namespace HugeCTR
